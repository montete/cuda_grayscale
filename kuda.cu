#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "io.h"
#include "kuda.h"
#include "lodepng.h"

void decodeTwoSteps(const char* filename, rgb_image *img)
{
  unsigned error;
  unsigned char* png;
  size_t pngsize;;
  
  lodepng_load_file(&png, &pngsize, filename);
  error = lodepng_decode32(&img->image, &img->width, &img->height, png, pngsize);
  if(error) printf("error %u: %s\n", error, lodepng_error_text(error));

  
}

void encodeOneStep(const char* filename, rgb_image *img)
{
  /*Encode the image*/
  unsigned error = lodepng_encode32_file(filename, img->image, img->width, img->height);

  /*if there's an error, display it*/
  if(error) printf("error %u: %s\n", error, lodepng_error_text(error));
}
void transformToGrayCuda(rgb_image *img){
	
	
	
	unsigned char* image = img->image;
    unsigned char* image_d;
    unsigned width = img->width;
    unsigned height = img->height;
    int N = (int)width * (int)height; 
    size_t size = N * 4 * sizeof(unsigned char);
	
    
	int device_count = 0;
	hipError_t status = hipGetDeviceCount(&device_count);
	
	status = hipMalloc((void **) &image_d, size);
	
	
	clock_t timer_start = clock();
	
	hipMemcpy(image_d, image,  size, hipMemcpyHostToDevice);
	
	clock_t timer_diff = clock() - timer_start;
	printf("CZas kopiowania RAM-CUDA: %gs\n", (timer_diff / (double)CLOCKS_PER_SEC));
	
	dim3 block_size(16, 16);
	dim3 num_blocks(img->width / block_size.x, img->height / block_size.y);
    setPixelToGrayscale<<<num_blocks, block_size>>>(image_d, img->width, img->height);
    
	timer_start = clock();
	
	hipMemcpy(image, image_d, size, hipMemcpyDeviceToHost);
	
	clock_t timer_diff2 = clock() - timer_start;
	
	printf("CZas kopiowania CUDA-RAM: %gs\n", (timer_diff2 / (double)CLOCKS_PER_SEC));
	hipFree(image_d);
	
	
}

__global__
void setPixelToGrayscale(unsigned char *image, unsigned width, unsigned height)
{
    float gray;
    float r, g, b;
	
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		r = image[4 * width * y + 4 * x + 0];
		g = image[4 * width * y + 4 * x + 1];
		b = image[4 * width * y + 4 * x + 2];
		gray = .299f*r + .587f*g + .114f*b;
		image[4 * width * y + 4 * x + 0] = gray;
		image[4 * width * y + 4 * x + 1] = gray;
		image[4 * width * y + 4 * x + 2] = gray;
		image[4 * width * y + 4 * x + 3] = 255;
	}
	
}


int main(int argc, char *argv[])
{
	const char* filename = argc > 1 ? argv[1] : "test.png";
	rgb_image img;
	
	decodeTwoSteps(filename, &img);
	transformToGrayCuda(&img);
	encodeOneStep("wynik3.png", &img);
	
	return 0;
}

